#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include"gradient_GPU.cuh"



__global__ void convX_gpu(float *img_in, int width, int height, int depth, float *filter, int t, float *outX)
{
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int Z = blockIdx.z*blockDim.z + threadIdx.z;
	int indice;
	float s = 0.0;

	if (X > t / 2 && X < width - t / 2 && Y >= 0 && Y < height && Z >= 0 && Z < depth) 
	{
		for (int j = -t / 2; j <= t / 2; j++)
		{
			indice = width*(height*Z + Y) + j + X;
			s += img_in[indice] * filter[t / 2 - j];
		}

		outX[width*(height*Z + Y) + X] = (float)s;
	}
}


__global__ void convY_gpu(float *img_in, int width, int height, int depth, float *filter, int t, float *outY)
{
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int Z = blockIdx.z*blockDim.z + threadIdx.z;
	int indice;
	float s = 0.0;

	if (Y > t / 2 && Y < height - t / 2 && X >= 0 && X < width && Z >= 0 && Z < depth)
	{
		for (int j = -t / 2; j <= t / 2; j++)
		{
			indice = width*(height*Z + j+Y) + X;
			s += img_in[indice] * filter[t / 2 - j];
		}

		outY[width*(height*Z + Y) + X] = (float)s;
	}
}
__global__ void convZ_gpu(float *img_in, int width, int height, int depth, float *filter, int t, float *outZ)
{
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int Z = blockIdx.z*blockDim.z + threadIdx.z;
	int indice;
	float s = 0.0;

	if (Z > t / 2 && Z < depth - t / 2 && Y >= 0 && Y < height && X >= 0 && X < width)
	{
		for (int j = -t / 2; j <= t / 2; j++)
		{
			indice = width*(height*(Z+j) + Y) + X;
			s += img_in[indice] * filter[t / 2 - j];
		}

		outZ[width*(height*Z + Y) + X] = (float)s;
	}
}



void gradient_gpu_main(float * img_in, int width, int height, int depth,
	float *filterSmooth, float *derivate, int t, float *gx, float*gy, float *gz)
{

	int nbThreadX = 10, nbThreadY = 10, nbThreadZ = 10;
	int size = width*height*depth;
	dim3 dimBlock(nbThreadX,nbThreadY,nbThreadZ);
	dim3 dimGrid((width + nbThreadX - 1) / nbThreadX, (height + nbThreadY - 1) / nbThreadY, (depth + nbThreadZ - 1) / nbThreadZ);
	float *img_in_cuda = NULL;
	float *filterSmooth_cuda = NULL;
	float *derivate_cuda = NULL;
	float *gx_cuda = NULL, *gy_cuda = NULL, *gz_cuda = NULL;
	float *tmp1, *tmp2;

	hipMalloc((void **)&img_in_cuda, size *sizeof(float));
	hipMemcpy(img_in_cuda, img_in, size * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void **)&filterSmooth_cuda, t* sizeof(float));
	hipMemcpy(filterSmooth_cuda, filterSmooth, t * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void **)&derivate_cuda, t * sizeof(float));
	hipMemcpy(derivate_cuda, derivate, t * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void **)&gx_cuda, size * sizeof(float));
	hipMalloc((void **)&gy_cuda, size * sizeof(float));
	hipMalloc((void **)&gz_cuda, size * sizeof(float));

	hipMalloc((void **)&tmp1, size * sizeof(float));
	hipMalloc((void **)&tmp2, size * sizeof(float));

	convZ_gpu <<<dimGrid, dimBlock >>> (img_in_cuda, width, height, depth, filterSmooth_cuda, t, tmp1);
	hipDeviceSynchronize();
	convY_gpu <<<dimGrid, dimBlock >>> (tmp1, width, height, depth, filterSmooth_cuda, t, tmp2);
	hipDeviceSynchronize();
	convX_gpu <<<dimGrid, dimBlock >>> (tmp2, width, height, depth, derivate_cuda, t, gx_cuda);
	hipDeviceSynchronize();
	
	convX_gpu <<<dimGrid, dimBlock >>> (tmp1, width, height, depth, filterSmooth_cuda, t, tmp2);
	hipDeviceSynchronize();
	convY_gpu <<<dimGrid, dimBlock >>> (tmp2, width, height, depth, derivate_cuda, t, gy_cuda);
	hipDeviceSynchronize();

	convX_gpu<<<dimGrid, dimBlock >>> (img_in_cuda, width, height, depth, filterSmooth_cuda, t, tmp1);
	hipDeviceSynchronize();
	convY_gpu<<<dimGrid, dimBlock >>> (tmp1, width, height, depth, filterSmooth_cuda, t, tmp2);
	hipDeviceSynchronize(); 
	convZ_gpu<<<dimGrid, dimBlock >>> (tmp2, width, height, depth, derivate_cuda, t, gz_cuda);
	hipDeviceSynchronize();

	
	hipMemcpy(gx_cuda, gx, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gy_cuda, gy, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gz_cuda, gz, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(gx_cuda);
	hipFree(gy_cuda);
	hipFree(gz_cuda);

	hipFree(img_in_cuda);
	hipFree(filterSmooth_cuda);
	hipFree(derivate_cuda);
	hipFree(tmp1);
	hipFree(tmp2);

}

 